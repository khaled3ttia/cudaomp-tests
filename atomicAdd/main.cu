#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void testAdd(int *x) {

  for (int i = 0; i < 10; i++) {

    atomicAdd(&x[i], 5);
  }
}

int main(void) {

  int N = 10;
  int *x, *d_x, *new_x;

  x = (int *)malloc(N * sizeof(int));

  new_x = (int *)malloc(N * sizeof(int));

  if (hipMalloc(&d_x, N * sizeof(int)) != hipSuccess) {
    printf("Error: failed to  allocate %lu bytes on device for d_x\n",
           N * sizeof(int));
  }

  for (int i = 0; i < N; i++) {

    x[i] = 0.0;
  }

  if (hipMemcpy(d_x, x, N * sizeof(int), hipMemcpyHostToDevice) !=
      hipSuccess) {
    printf("Error: failed to copy %lu bytes from host to device\n",
           N * sizeof(int));
  }

  testAdd<<<1, 10>>>(d_x);

  if (hipMemcpy(new_x, d_x, N * sizeof(int), hipMemcpyDeviceToHost) !=
      hipSuccess) {
    printf("Error: failed to copy %lu bytes from device to host\n",
           N * sizeof(int));
  }

  for (int i = 0; i < N; i++) {

    printf("%d ", new_x[i]);
  }

  if (hipFree(d_x) != hipSuccess) {
    printf("Error: failed to free memory from the device\n");
  }

  free(new_x);
  free(x);
}
