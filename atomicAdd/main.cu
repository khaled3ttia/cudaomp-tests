#include <iostream>
#include <hip/hip_runtime.h>

__global__ void testAdd(int *x){


    for (int i = 0 ; i < 10 ; i++){

        atomicAdd(&x[i], 5);


    }

}

int main(void){

    int N = 10;
    int *x, *d_x, *new_x;

   
    x = (int*)malloc(N*sizeof(int));

    new_x = (int*)malloc(N*sizeof(int));


    hipMalloc(&d_x, N*sizeof(int));

    for (int i = 0 ; i < N; i++){

        x[i] = 0.0;
    }

    hipMemcpy(d_x, x, N*sizeof(int), hipMemcpyHostToDevice);

    testAdd<<<1, 10>>>(d_x);

    
    hipMemcpy(new_x, d_x, N*sizeof(int), hipMemcpyDeviceToHost);

    
    for (int i = 0; i < N; i++){

        printf("%d ", new_x[i]); 
    }
    

    hipFree(d_x);
    free(new_x);
    free(x);
    

}
