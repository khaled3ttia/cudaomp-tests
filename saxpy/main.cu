#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void saxpy(int n, double a, double *x, double *y) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {

    y[i] = a * x[i] + y[i];
  }
}

int main(void) {

  int N = 1 << 20;
  printf("N is %d\n", N);

  double *x, *y, *d_x, *d_y;
  x = (double *)malloc(N * sizeof(double));
  y = (double *)malloc(N * sizeof(double));

  if (hipMalloc(&d_x, N * sizeof(double)) != hipSuccess) {

    printf("[Error] failed to allocate %lu bytes for d_x on device\n",
           N * sizeof(double));
  }

  if (hipMalloc(&d_y, N * sizeof(double)) != hipSuccess) {

    printf("[Error] failed to allocate %lu bytes for d_y on device\n",
           N * sizeof(double));
  }

  for (int i = 0; i < N; ++i) {

    x[i] = 1.0;
    y[i] = 2.0;
  }

  if (hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice) !=
      hipSuccess) {
    printf("[Error] failed to copy %lu bytes from x to d_x on device\n",
           N * sizeof(double));
  }

  if (hipMemcpy(d_y, y, N * sizeof(double), hipMemcpyHostToDevice) !=
      hipSuccess) {
    printf("[Error] failed to copy %lu bytes from y to d_y on device\n",
           N * sizeof(double));
  }

  saxpy<<<(N + 255) / 256, 256>>>(N, 2.0, d_x, d_y);

  if (hipMemcpy(y, d_y, N * sizeof(double), hipMemcpyDeviceToHost) !=
      hipSuccess) {

    printf("[Error] failed to copy %lu bytes from d_y (device) to y (host)\n",
           N * sizeof(double));
  }

  double maxError = 0.0;
  for (int i = 0; i < N; i++) {
    if (abs(y[i] - 4.0) > maxError) {
      maxError = abs(y[i] - 4.0);
    }
  }
  printf("Max error %f\n", maxError);

  if (hipFree(d_x) != hipSuccess) {

    printf("[Error] failed to free d_x on device\n");
  }
  if (hipFree(d_y) != hipSuccess) {

    printf("[Error] failed to free d_y on device\n");
  }
  free(x);
  free(y);
}
